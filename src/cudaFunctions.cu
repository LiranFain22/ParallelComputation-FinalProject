#include "hip/hip_runtime.h"
#include "cudaFunctions.h"
#include <math.h>

__device__ int calcDiff(int p, int o)
{
    return abs((p - o) / p);
}

__global__ void findMatch(Picture* picture, Obj* object, int matchingValue, int row, int col, int picSize, int objSize, int* isMatch)
{
    int result = 0;
    if ((row + object->objSize) < picture->picSize &&
        (col + object->objSize) < picture->picSize)
    {
            for(int i = 0; i < object->objSize; i++)
            {
                for(int j = 0; j < object->objSize; j++)
                {
                    int objIdx = (i * object->objSize) + j;
                    int picIdx = ((row + i) * picture->picSize) + (col + j);
                    result += calcDiff(picture->picArr[picIdx], object->objArr[objIdx]);
                    if ( result > matchingValue)
                    {
                        *isMatch = 0;
                        return;
                    }
                }
            }
            *isMatch = 1;
    }
    else
    {
        *isMatch = 0;
    }
}

void cudaFuncs(Picture* picture, Obj* object, int* matchingValue)
{
    Picture *dev_pic = 0;
    Obj *dev_object = 0;
    hipError_t status = hipSuccess;
    int numOfThreads, numOfBlocks;
    int pictureSize = picture->picSize;
    int objectSize = object->objSize;

    if ((picture->picSize * picture->picSize) > MAX_THREADS_IN_BLOCK)
    {
        numOfThreads = MAX_THREADS_IN_BLOCK;
        numOfBlocks = ((picture->picSize * picture->picSize)/numOfThreads) + 1;
    }
    else
    {
        numOfThreads = (picture->picSize * picture->picSize);
        numOfBlocks = 1;
    }

    status = hipMalloc((void**)&dev_pic, sizeof(int) * pictureSize);
    if(status != hipSuccess)
    {
        printf("Failded to allocate memory for picture in GPU\n");
        exit(1);
    }
    status = hipMemcpy(dev_seq1, seq1, seq1Len*sizeof(char),hipMemcpyHostToDevice);
    checkStatus(status , "CudaMemcpy to DEVICE Failed! (dev_seq1)");

    //Seq2
    status = hipMalloc((void**)&dev_seq2, sizeof(char) * seq2Len);
    checkStatus(status , "CudaMalloc Failed! (dev_seq2)");
    status = hipMemcpy(dev_seq2, seq2, seq2Len*sizeof(char),hipMemcpyHostToDevice);
    checkStatus(status , "CudaMemcpy to DEVICE Failed! (dev_seq2)");

    //Weight array
    status = hipMalloc((void**)&dev_weights, sizeof(float) * WEIGHTS);
    checkStatus(status , "CudaMalloc Failed! (dev_weights)");
    status = hipMemcpy(dev_weights, weightArr, WEIGHTS*sizeof(float),hipMemcpyHostToDevice);
    checkStatus(status , "CudaMemcpy to DEVICE Failed! (dev_weights)");

    //Device best score
    status = hipMalloc((void**)&dev_bestMutant, sizeof(Score));
    checkStatus(status , "CudaMalloc Failed! (dev_bestMutant)tell me why??");
    status = hipMemcpy(dev_bestMutant, &tempScore, sizeof(Score),hipMemcpyHostToDevice);
    checkStatus(status , "CudaMemcpy to DEVICE Failed! (dev_weights)");

    //Device score array
    status = hipMalloc((void**)&dev_mutantArr, sizeof(Score) * (*mutantArrSize));
    checkStatus(status , "CudaMalloc Failed! (dev_bestMutant)");
    status = hipMemcpy(dev_mutantArr, mutantArr, sizeof(Score) * (*mutantArrSize),hipMemcpyHostToDevice);
    checkStatus(status , "CudaMemcpy to DEVICE Failed! (dev_weights)");

    //---------------- START CUDA -----------------------
    cudaCalculations<<<numOfBlocks, numOfThreads>>>(dev_seq1, dev_seq2, seq2Len, dev_mutantArr, *mutantArrSize, dev_weights, dev_bestMutant);
    status = hipDeviceSynchronize();
    checkStatus(status , "Synchronize Failed!");

    //---------------- COPY DATA BACK TO HOST -----------------------
    status = hipMemcpy(bestMutant, dev_bestMutant, sizeof(Score),hipMemcpyDeviceToHost);
    checkStatus(status , "CudaMemcpy to DEVICE Failed! (bestMutant)");

    //---------------- FREE MEMORY -----------------------
    status = hipFree(dev_seq1);
    checkStatus(status,"Cuda Free Failed! (dev_seq1)");
    status = hipFree(dev_seq2);
    checkStatus(status,"Cuda Free Failed! (dev_seq2)");
    status = hipFree(dev_weights);
    checkStatus(status,"Cuda Free Failed! (dev_weights)");
    status = hipFree(dev_bestMutant);
    checkStatus(status,"Cuda Free Failed! (dev_bestMutant)");
    status = hipFree(dev_mutantArr);
    checkStatus(status,"Cuda Free Failed! (dev_mutantArr)");
}