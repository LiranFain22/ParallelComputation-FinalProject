#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h" 
#include "helper_functions.h" 
#include "cudaFunctions.h"
#include <math.h>

void checkStatus(hipError_t cudaStatus, const char* errorMsg)
{
    if(cudaStatus != hipSuccess)
    {
        printf("%s\n",errorMsg);
        exit(1);
    }
}

__device__ int calcDiff(int p, int o)
{
    return abs((p - o) / p);
}

__global__ void findMatch(int* picture, int* object, int matchingValue, int picSize, int objSize, Match* match, int objectId)
{
    printf("start findMatch function\n\n");
    int result = 0;

    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int s = bx * blockDim.x + tx;
    int foundMatch = 1;
    __shared__ int bestMatchIdx;
    bestMatchIdx = -1;

    int row = s / picSize;
    int col = s - picSize * row;

    printf("row = %d, col = %d\n", row, col);

    if ((row + objSize) < picSize && (col + objSize) < picSize)
    {
            for(int i = 0; i < objSize; i++)
            {
                for(int j = 0; j < objSize; j++)
                {
                    int objIdx = (i * objSize) + j;
                    int picIdx = ((row + i) * picSize) + (col + j);
                    result += calcDiff(picture[picIdx], object[objIdx]);
                    if (result > matchingValue || match->isMatch)
                    {
                        foundMatch = 0;
                        break;
                    }
                }
                if(foundMatch == 0) break;
            }
            __syncthreads();
            //atomic min if foundMatch is 1
            if (foundMatch == 1) 
            {
                atomicMin(&bestMatchIdx, s);
            }
            __syncthreads();
            // check if i am min
                // if i do, update is match with row, col, obj id, is match
            if (s == bestMatchIdx)
            {
                (*match).isMatch = 1;
                (*match).row = row;
                (*match).col = col;
                (*match).objectId = objectId;
            }
    }
}


void cudaFuncs(Picture* picture, Obj* object, int* matchingValue, Match* match)
{
    printf("start cudaFuncs function\n\n");
    int *dev_pic = 0;
    int *dev_obj = 0;
    hipError_t status = hipSuccess;
    int numOfThreads, numOfBlocks;
    int pictureSize = picture->picSize;
    int objectSize = object->objSize;
    int objId = object->objId;
    Match* dev_match = 0;

    if ((pictureSize * pictureSize) > MAX_THREADS_IN_BLOCK)
    {
        numOfThreads = MAX_THREADS_IN_BLOCK;
        numOfBlocks = ((pictureSize * pictureSize)/numOfThreads) + 1;
    }
    else
    {
        numOfThreads = (pictureSize * pictureSize);
        numOfBlocks = 1;
    }

    printf("start allocating memory\n");
    printf("picSize for allocation is %d\n", pictureSize);
    // picture's device
    status = hipMalloc((void**)&dev_pic, sizeof(int) * pictureSize * pictureSize);
    checkStatus(status, "Faild to allocate memory for picture in GPU\n");

    printf("succeeded allocating memory for dev_pic\n");

    status = hipMemcpy(dev_pic, picture->picArr, pictureSize*pictureSize*sizeof(int),hipMemcpyHostToDevice);
    checkStatus(status, "CudaMemcpy to device failed! (dev_pic)\n");

    printf("succeeded copy memory for dev_pic\n");

    // object's device
    status = hipMalloc((void**)&dev_obj, sizeof(int) * objectSize * objectSize);
    checkStatus(status, "Faild to allocate memory for object in GPU\n");

    printf("succeeded allocating memory for dev_obj\n");

    status = hipMemcpy(dev_obj, object->objArr, objectSize*objectSize*sizeof(int),hipMemcpyHostToDevice);
    checkStatus(status, "CudaMemcpy to device failed! (dev_obj)\n");

    printf("succeeded copy memory for dev_obj\n");

    // match's device
    status = hipMalloc((void**)&dev_match, sizeof(Match));
    checkStatus(status, "Faild to allocate memory for match in GPU\n");

    printf("succeeded allocating memory for dev_match\n");

    status = hipMemcpy(dev_match, match, sizeof(Match),hipMemcpyHostToDevice);
    checkStatus(status, "CudaMemcpy to device failed! (dev_match)\n");

    printf("succeeded copy memory for dev_match\n");

    // starting CUDA
    findMatch<<<numOfBlocks, numOfThreads>>>(dev_pic, dev_obj, *matchingValue, pictureSize, objectSize, dev_match, objId);
    
    
    status = hipDeviceSynchronize();
    checkStatus(status, "Synchronize Failed!\n");

    // //---------------- COPY DATA BACK TO HOST -----------------------
    // status = hipMemcpy(bestMutant, dev_bestMutant, sizeof(Score),hipMemcpyDeviceToHost);
    // checkStatus(status , "CudaMemcpy to device failed! (bestMutant)");

    // copy data back to host
    status = hipMemcpy(match, dev_match, sizeof(Match),hipMemcpyDeviceToHost);
    checkStatus(status, "CudaMemcpy to host failed! (isMatch)\n");

    // free memory
    status = hipFree(dev_pic);
    checkStatus(status,"Cuda Free function failed! (dev_pic)\n");
    status = hipFree(dev_obj);
    checkStatus(status,"Cuda Free function failed! (dev_obj)\n");
    status = hipFree(dev_match);
    checkStatus(status,"Cuda Free function failed! (dev_match)\n");
}